#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 4096  // 矩阵大小
#define BLOCK_SIZE 32  // 每个线程块的大小

// GPU 核函数：使用共享内存优化的矩阵乘法
__global__ void matmul_shared(const float *A, const float *B, float *C, int n) {
    __shared__ float Asub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bsub[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y; // 当前线程对应的行
    int col = blockIdx.x * blockDim.x + threadIdx.x; // 当前线程对应的列

    float sum = 0.0f;

    for (int t = 0; t < (n + BLOCK_SIZE - 1) / BLOCK_SIZE; ++t) {
        // 加载 A 和 B 的子块到共享内存
        if (row < n && t * BLOCK_SIZE + threadIdx.x < n) {
            Asub[threadIdx.y][threadIdx.x] = A[row * n + t * BLOCK_SIZE + threadIdx.x];
        } else {
            Asub[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < n && t * BLOCK_SIZE + threadIdx.y < n) {
            Bsub[threadIdx.y][threadIdx.x] = B[(t * BLOCK_SIZE + threadIdx.y) * n + col];
        } else {
            Bsub[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads(); // 确保所有线程加载完成

        // 计算子块的结果
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            sum += Asub[threadIdx.y][k] * Bsub[k][threadIdx.x];
        }

        __syncthreads(); // 确保所有线程完成计算
    }

    // 写入结果矩阵
    if (row < n && col < n) {
        C[row * n + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(float);

    // 分配主机内存
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // 初始化矩阵 A 和 B
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = (float)(rand() % 100);
        h_B[i] = (float)(rand() % 100);
    }

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 定义线程块和网格大小
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE); // 每个线程块 BLOCK_SIZE x BLOCK_SIZE 个线程
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    // 启动计时器
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // 启动 GPU 核函数
    matmul_shared<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    // 停止计时器
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Matrix multiplication took %.6f seconds.\n", elapsedTime/1000.0f);

    // 将结果从设备复制回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // 释放主机内存
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}